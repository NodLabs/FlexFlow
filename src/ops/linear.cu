#include "hip/hip_runtime.h"
/* Copyright 2019 Stanford, NVIDIA
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "model.h"
#include "cuda_helper.h"

Tensor FFModel::dense(std::string name,
                      const Tensor& input,
                      int outDim, 
                      ActiMode activation,
                      bool use_bias, 
                      Initializer* kernel_initializer,
                      Initializer* bias_initializer)
{
  if (kernel_initializer == NULL) {
    int seed = std::rand();
    kernel_initializer = new GlorotUniform(seed);
  }
  if (bias_initializer == NULL) {
    bias_initializer = new ZeroInitializer();
  }
  Linear *li = new Linear(*this, name, input, outDim, activation, use_bias,
                          kernel_initializer, bias_initializer);
  li->add_to_model(*this);
  return li->outputs[0];
}

Linear* FFModel::dense(std::string name,
                       int inDim,
                       int outDim, 
                       ActiMode activation,
                       bool use_bias, 
                       Initializer* kernel_initializer,
                       Initializer* bias_initializer)
{
  if (kernel_initializer == NULL) {
    int seed = std::rand();
    kernel_initializer = new GlorotUniform(seed);
  }
  if (bias_initializer == NULL) {
    bias_initializer = new ZeroInitializer();
  }
  Linear *li = new Linear(*this, name, inDim, outDim, activation, use_bias,
                          kernel_initializer, bias_initializer);
  return li;
}

// Deprecated API -- TO BE REMOVED
Tensor FFModel::linear(std::string name,
                       const Tensor& input,
                       int outDim,
                       ActiMode activation,
                       bool use_bias,
                       Initializer* kernel_initializer,
                       Initializer* bias_initializer)
{
  fprintf(stderr, "FFModel::linear is deprecated and will be removed,"
         "please use FFModel::dense instead");
  return dense(name, input, outDim, activation,
               kernel_initializer, bias_initializer);
}

Linear::Linear(FFModel& model,
               const std::string& pcname,
               const Tensor& _input,
               int out_dim,
               ActiMode _activation,
               bool use_bias,
               Initializer* kernel_initializer,
               Initializer* bias_initializer)
: Op(pcname, _input), 
  in_channels(_input.adim[0]), out_channels(out_dim),
  activation(_activation),
  profiling(model.config.profiling)
{
  assert(_input.numDim == 2);
  create_kernel_bias(model, use_bias, kernel_initializer, bias_initializer);
  create_output_and_partition(model);
}

Linear::Linear(FFModel& model,
               const std::string& pcname,
               int in_dim,
               int out_dim,
               ActiMode _activation,
               bool use_bias,
               Initializer* kernel_initializer,
               Initializer* bias_initializer)
: Op(pcname, 1), 
  in_channels(in_dim), out_channels(out_dim),
  activation(_activation),
  profiling(model.config.profiling)
{
  create_kernel_bias(model, use_bias, kernel_initializer, bias_initializer);
}

Tensor Linear::init_inout(FFModel& model, const Tensor& _input)
{
  add_to_model(model);
  assert(_input.numDim == 2);
  assert(_input.adim[0] == in_channels);
  inputs[0] = _input;
  create_output_and_partition(model);
  return outputs[0];
}

void Linear::add_to_model(FFModel& model)
{
  model.layers.push_back(this);
  model.parameters.push_back(weights[0]);
  if (numWeights > 1) { // bias is used
    assert(numWeights == 2);
    model.parameters.push_back(weights[1]);
  }
}

void Linear::create_kernel_bias(FFModel& model, bool use_bias, Initializer* kernel_initializer, Initializer* bias_initializer)
{
  // Retrive the task indexspace for the op
  std::string pcname = name;
  task_is = IndexSpaceT<2>(model.get_or_create_task_is(2, pcname));

  // Create kernel tensor
  {
    const int dims[2] = {out_channels, in_channels};
    weights[numWeights++] = model.create_linear_weight<2>(this, dims, (IndexSpaceT<2>)task_is, DT_FLOAT, kernel_initializer);
  }
  // Create bias tensor
  if (use_bias) {
    const int dims[1] = {out_channels};
    weights[numWeights++] = model.create_linear_weight<1>(this, dims, (IndexSpaceT<2>)task_is, DT_FLOAT, bias_initializer);
  }
}

void Linear::create_output_and_partition(FFModel& model)
{
  // Retrive the task indexspace for the op
  std::string pcname = name;
  task_is = IndexSpaceT<2>(model.get_or_create_task_is(2, pcname));

  Context ctx = model.config.lg_ctx;
  Runtime* runtime = model.config.lg_hlr;
  Rect<2> part_rect = runtime->get_index_space_domain(ctx, task_is);
  int num_par_c = part_rect.hi[0] - part_rect.lo[0] + 1;
  int num_par_n = part_rect.hi[1] - part_rect.lo[1] + 1;
  int in_dim = inputs[0].adim[0];
  assert(in_dim == in_channels);
  int batch_size = inputs[0].adim[1];
  {
    const int dims[2] = {batch_size, out_channels};
    outputs[0] = model.create_tensor<2>(dims, (IndexSpaceT<2>)task_is, DT_FLOAT);
  }
  // Compute partition bound for input
  Rect<2> input_rect = runtime->get_index_partition_color_space(
      ctx, inputs[0].part.get_index_partition());
  // Create replica tensor
  if (num_par_c > 1) {
    const int dims[3] = {num_par_c, batch_size, in_dim};
    replica = model.create_linear_replica<3>(dims, (IndexSpaceT<2>)task_is, DT_FLOAT);
    {
      Rect<2> extent(Point<2>(0, 0), Point<2>(in_dim-1, batch_size/num_par_n-1));
      Transform<2, 2> transform;
      transform[0][0] = 0;
      transform[0][1] = 0;
      transform[1][0] = 0;
      transform[1][1] = batch_size/num_par_n;
      IndexPartition ip = runtime->create_partition_by_restriction(
          ctx, inputs[0].region.get_index_space(), task_is, transform, extent);
      input_lps[0] = runtime->get_logical_partition(
          ctx, inputs[0].region, ip);
    }
    // Backward use the same ip as inputs[0]
    input_grad_lps[0] = inputs[0].part_grad;
    {
      IndexSpaceT<2> input_task_is = IndexSpaceT<2>(model.get_or_create_task_is(input_rect));
      const coord_t num_parts[2] = {input_rect.hi[0] - input_rect.lo[0] + 1,
                                    input_rect.hi[1] - input_rect.lo[1] + 1};
      Rect<3> extent(Point<3>(0, 0, 0),
          Point<3>(in_dim/num_parts[0]-1, batch_size/num_parts[1]-1, num_par_c-1));
      Transform<3, 2> transform;
      for (int i = 0; i < 3; i++)
        for (int j = 0; j < 2; j++)
          transform[i][j] = 0;
      transform[0][0] = in_dim / num_parts[0];
      transform[1][1] = batch_size / num_parts[1];
      IndexPartition ip = runtime->create_partition_by_restriction(
          ctx, replica.region_grad.get_index_space(), input_task_is,
          transform, extent);
      assert(runtime->is_index_partition_disjoint(ctx, ip));
      assert(runtime->is_index_partition_complete(ctx, ip));
      // Note we use replica.part to save how to partition the replica
      // to compute input_grad_lps
      replica.part = runtime->get_logical_partition(
          ctx, replica.region_grad, ip);
    }
  } else {
    if (input_rect == part_rect) {
      input_lps[0] = inputs[0].part;
      input_grad_lps[0] = inputs[0].part_grad;
    } else {
      Rect<2> extent(Point<2>(0,0), Point<2>(in_dim-1,batch_size/num_par_n-1));
      Transform<2, 2> transform;
      transform[0][0] = 0;
      transform[0][1] = 0;
      transform[1][0] = 0;
      transform[1][1] = batch_size / num_par_n;
      IndexPartition ip = runtime->create_partition_by_restriction(
          ctx, inputs[0].region.get_index_space(), task_is, transform, extent);
      assert(runtime->is_index_partition_disjoint(ctx, ip));
      assert(runtime->is_index_partition_complete(ctx, ip));
      input_lps[0] = runtime->get_logical_partition(
          ctx, inputs[0].region, ip);
      input_grad_lps[0] = runtime->get_logical_partition(
          ctx, inputs[0].region_grad, ip);
    }
  }
}


/*
  regions[0](O): output
  regions[1](I): kernel
  regions[2](I): bias
*/
OpMeta* Linear::init_task(const Task *task,
                          const std::vector<PhysicalRegion> &regions,
                          Context ctx, Runtime *runtime)
{
  assert(regions.size() == 3);
  assert(task->regions.size() == 3);
  const Linear* linear = (Linear*) task->args;
  FFHandler handle = *((const FFHandler*) task->local_args);
  //TensorAccessorR<float, 2> acc_input(
  //    regions[0], task->regions[0], FID_DATA, ctx, runtime);
  TensorAccessorW<float, 2> acc_output(
      regions[0], task->regions[0], FID_DATA, ctx, runtime,
      false/*readOutput*/);
  TensorAccessorR<float, 2> acc_kernel(
      regions[1], task->regions[1], FID_DATA, ctx, runtime);
  TensorAccessorR<float, 1> acc_bias(
      regions[2], task->regions[2], FID_DATA, ctx, runtime);
  //int in_dim = acc_input.rect.hi[0] - acc_input.rect.lo[0] + 1;
  int in_dim = acc_kernel.rect.hi[0] - acc_kernel.rect.lo[0] + 1;
  int out_dim = acc_output.rect.hi[0] - acc_output.rect.lo[0] + 1;
  int batch_size = acc_output.rect.hi[1] - acc_output.rect.lo[1] + 1;
  printf("init linear (input): in_dim(%d) out_dim(%d) batch_size(%d)\n",
      in_dim, out_dim, batch_size);
  LinearMeta* m = new LinearMeta(handle);

  float* dram_one_ptr = (float *) malloc(sizeof(float) * batch_size);
  for (int i = 0; i < batch_size; i++)
    dram_one_ptr[i] = 1.0f;
  float* fb_one_ptr;
  checkCUDA(hipMalloc(&fb_one_ptr, sizeof(float) * batch_size));
  checkCUDA(hipMemcpy(fb_one_ptr, dram_one_ptr,
                       sizeof(float) * batch_size, hipMemcpyHostToDevice));
  m->one_ptr = (const float*) fb_one_ptr;
  if (linear->activation != AC_MODE_NONE) {
    hipdnnActivationMode_t mode;
    switch (linear->activation) {
      case AC_MODE_RELU:
        mode = HIPDNN_ACTIVATION_RELU;
        break;
      case AC_MODE_SIGMOID:
        mode = HIPDNN_ACTIVATION_SIGMOID;
        break;
      default:
        // Unsupported activation mode
        assert(false);
    }
    checkCUDNN(hipdnnCreateActivationDescriptor(&m->actiDesc));
    checkCUDNN(hipdnnSetActivationDescriptor(m->actiDesc, mode,
                                            HIPDNN_PROPAGATE_NAN, 0.0));
    checkCUDNN(hipdnnCreateTensorDescriptor(&m->outputTensor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(m->outputTensor,
                                          HIPDNN_TENSOR_NCHW,
                                          HIPDNN_DATA_FLOAT,
                                          batch_size, out_dim, 1, 1));
  }
  return m;
}

void Linear::init(const FFModel& ff)
{
	std::cout << "linear layer" << std::endl;
  ArgumentMap argmap;
  Context ctx = ff.config.lg_ctx;
  Runtime* runtime = ff.config.lg_hlr;
  Rect<2> rect = runtime->get_index_space_domain(ctx, task_is);
  int idx = 0;
  printf("start setup handlers\n");
  for (PointInRectIterator<2> it(rect); it(); it++) {
    FFHandler handle = ff.handlers[idx++];
    argmap.set_point(*it, TaskArgument(&handle, sizeof(FFHandler)));
  }
  printf("launcher setup\n");
  IndexLauncher launcher(LINEAR_INIT_TASK_ID, task_is,
                         TaskArgument(this, sizeof(Linear)), argmap,
                         Predicate::TRUE_PRED, false/*must*/, 0/*mapper_id*/,
                         FFConfig::get_hash_id(std::string(name)));
  //launcher.add_region_requirement(
  //    RegionRequirement(input_lps[0], 0/*projection id*/,
  //                      READ_ONLY, EXCLUSIVE, inputs[0].region));
  //launcher.add_field(0, FID_DATA);
  printf("requirement 1\n");
  launcher.add_region_requirement(
      RegionRequirement(outputs[0].part, 0/*projection id*/,
                        WRITE_ONLY, EXCLUSIVE, outputs[0].region));
  launcher.add_field(0, FID_DATA);
  printf("requirement 2\n");
  launcher.add_region_requirement(
      RegionRequirement(weights[0].part, 0/*projection id*/,
                        READ_ONLY, EXCLUSIVE, weights[0].region));
  launcher.add_field(1, FID_DATA);
  printf("requirement 3\n");
  launcher.add_region_requirement(
      RegionRequirement(weights[1].part, 0/*projection id*/,
                        READ_ONLY, EXCLUSIVE, weights[1].region));
  launcher.add_field(2, FID_DATA);
  printf("execute\n");
  FutureMap fm = runtime->execute_index_space(ctx, launcher);
  fm.wait_all_results();
  idx = 0;
  for (PointInRectIterator<2> it(rect); it(); it++) {
    meta[idx++] = fm.get_result<OpMeta*>(*it);
  }
  printf("get result endup\n");
}

/*
  regions[0](I); input
  regions[1](O): output
  regions[2](I): kernel
  regions[3](I): bias
*/
__host__
void Linear::forward_task(const Task *task,
                          const std::vector<PhysicalRegion> &regions,
                          Context ctx, Runtime *runtime)
{
  assert(regions.size() == 4);
  assert(task->regions.size() == 4);
  float alpha = 1.0f, beta = 0.0f;
  const Linear* linear = (Linear*) task->args;
  const LinearMeta* m = *((LinearMeta**) task->local_args);
  TensorAccessorR<float, 2> acc_input(
      regions[0], task->regions[0], FID_DATA, ctx, runtime);
  TensorAccessorW<float, 2> acc_output(
      regions[1], task->regions[1], FID_DATA, ctx, runtime,
      false/*readOutput*/);
  TensorAccessorR<float, 2> acc_kernel(
      regions[2], task->regions[2], FID_DATA, ctx, runtime);
  TensorAccessorR<float, 1> acc_bias(
      regions[3], task->regions[3], FID_DATA, ctx, runtime);
  int in_dim = acc_input.rect.hi[0] - acc_input.rect.lo[0] + 1;
  int out_dim = acc_output.rect.hi[0] - acc_output.rect.lo[0] + 1;
  int batch_size = acc_input.rect.hi[1] - acc_input.rect.lo[1] + 1;
  assert(acc_output.rect.volume() == out_dim * batch_size);
  assert(acc_kernel.rect.volume() == in_dim * out_dim);
  assert(acc_bias.rect.volume() == out_dim);

  hipEvent_t t_start, t_end;
  if (linear->profiling) {
    hipEventCreate(&t_start);
    hipEventCreate(&t_end);
    hipEventRecord(t_start);
  }
#ifndef DISABLE_LEGION_CUDA_HIJACK
  std::cout << "create stream" << std::endl;
  hipStream_t stream;
    std::cout << "hipStreamCreate stream" << std::endl;
  checkCUDA(hipStreamCreate(&stream));
    std::cout << "hipblasSetStream handle blas" << std::endl;
  checkCUDA(hipblasSetStream(m->handle.blas, stream));
    std::cout << "hipdnnSetStream handle dnn" << std::endl;
  checkCUDNN(hipdnnSetStream(m->handle.dnn, stream));
#endif
  checkCUDA(hipblasSgemm(m->handle.blas, HIPBLAS_OP_T, HIPBLAS_OP_N,
                        out_dim, batch_size, in_dim,
                        &alpha, acc_kernel.ptr, in_dim,
                        acc_input.ptr, in_dim, &beta,
                        acc_output.ptr, out_dim));
  checkCUDA(hipblasSgemm(m->handle.blas, HIPBLAS_OP_T, HIPBLAS_OP_N,
                        out_dim, batch_size, 1,
                        &alpha, acc_bias.ptr, 1,
                        m->one_ptr, 1, &alpha,
                        acc_output.ptr, out_dim));
  if (linear->activation != AC_MODE_NONE) {
    checkCUDNN(hipdnnActivationForward(m->handle.dnn, m->actiDesc,
        &alpha, m->outputTensor, acc_output.ptr,
        &beta, m->outputTensor, acc_output.ptr));
  }
  if (linear->profiling) {
    hipEventRecord(t_end);
    checkCUDA(hipEventSynchronize(t_end));
    float elapsed = 0;
    checkCUDA(hipEventElapsedTime(&elapsed, t_start, t_end));
    hipEventDestroy(t_start);
    hipEventDestroy(t_end);
    printf("Linear forward time = %.2lfms\n", elapsed);
    //print_tensor<2, float>(acc_input.ptr, acc_input.rect, "[Linear:forward:input]");
    //print_tensor<2, float>(acc_kernel.ptr, acc_kernel.rect, "[Linear:forward:kernel]");
    //print_tensor<1, float>(acc_bias.ptr, acc_bias.rect, "[Linear:forward:bias]");
    //print_tensor<2, float>(acc_output.ptr, acc_output.rect, "[Linear:forward:output]");
    checkCUDA(hipDeviceSynchronize());
  }
}

void Linear::forward(const FFModel& ff)
{
  ArgumentMap argmap;
  Context ctx = ff.config.lg_ctx;
  Runtime* runtime = ff.config.lg_hlr;
  Rect<2> rect = runtime->get_index_space_domain(ctx, task_is);
  int idx = 0;
  for (PointInRectIterator<2> it(rect); it(); it++) {
    OpMeta* mp = meta[idx++];
    argmap.set_point(*it, TaskArgument(&mp, sizeof(OpMeta*)));
  }
  IndexLauncher launcher(LINEAR_FWD_TASK_ID, task_is,
                         TaskArgument(this, sizeof(Linear)), argmap,
                         Predicate::TRUE_PRED, false/*must*/, 0/*mapper_id*/,
                         FFConfig::get_hash_id(std::string(name)));
  launcher.add_region_requirement(
      RegionRequirement(input_lps[0], 0/*projection id*/,
                        READ_ONLY, EXCLUSIVE, inputs[0].region));
  launcher.add_field(0, FID_DATA);
  launcher.add_region_requirement(
      RegionRequirement(outputs[0].part, 0/*projection id*/,
                        WRITE_ONLY, EXCLUSIVE, outputs[0].region));
  launcher.add_field(1, FID_DATA);
  launcher.add_region_requirement(
      RegionRequirement(weights[0].part, 0/*projection id*/,
                        READ_ONLY, EXCLUSIVE, weights[0].region));
  launcher.add_field(2, FID_DATA);
  launcher.add_region_requirement(
      RegionRequirement(weights[1].part, 0/*projection id*/,
                        READ_ONLY, EXCLUSIVE, weights[1].region));
  launcher.add_field(3, FID_DATA);
  runtime->execute_index_space(ctx, launcher);
}

__global__
void sigmoid_backward(float *grad_ptr, const float *output, int n)
{
  CUDA_KERNEL_LOOP(i, n)
  {
    grad_ptr[i] = grad_ptr[i] * output[i] * (1 - output[i]);
  }
}

/*
  regions[0](I): input
  regions[1](I/O): replica_grad or input_grad
  regions[2](I): output
  regions[3](I/O): output_grad
  regions[4](I): filter
  regions[5](I/O): filter_grad
  regions[6](I/O): bias_grad
*/
__host__
void Linear::backward_task(const Task *task,
                           const std::vector<PhysicalRegion> &regions,
                           Context ctx, Runtime *runtime)
{
  assert(regions.size() == 7);
  assert(task->regions.size() == 7);
  float alpha = 1.0f;
  const Linear* linear = (Linear*) task->args;
  const LinearMeta* m = *((LinearMeta**) task->local_args);
  float* input_grad = NULL;
  TensorAccessorR<float, 2> acc_input(
      regions[0], task->regions[0], FID_DATA, ctx, runtime);
  TensorAccessorR<float, 2> acc_output(
      regions[2], task->regions[2], FID_DATA, ctx, runtime);
  int in_dim = acc_input.rect.hi[0] - acc_input.rect.lo[0] + 1;
  int batch_size = acc_input.rect.hi[1] - acc_input.rect.lo[1] + 1;
  int out_dim = acc_output.rect.hi[0] - acc_output.rect.lo[0] + 1;
  Domain domain = runtime->get_index_space_domain(
      ctx, task->regions[1].region.get_index_space());
  if (domain.get_dim() == 3) {
    TensorAccessorW<float, 3> acc_replica_grad(
        regions[1], task->regions[1], FID_DATA, ctx, runtime,
        true/*readOutput*/);
    assert(acc_replica_grad.rect.volume() == in_dim * batch_size);
    input_grad = acc_replica_grad.ptr;
  } else {
    TensorAccessorW<float, 2> acc_replica_grad(
        regions[1], task->regions[1], FID_DATA, ctx, runtime,
        true/*readOutput*/);
    assert(acc_replica_grad.rect.volume() == in_dim * batch_size);
    input_grad = acc_replica_grad.ptr;
  }
  TensorAccessorW<float, 2> acc_output_grad(
      regions[3], task->regions[3], FID_DATA, ctx, runtime,
      true/*readOutput*/);
  TensorAccessorR<float, 2> acc_kernel(
      regions[4], task->regions[4], FID_DATA, ctx, runtime);
  TensorAccessorW<float, 2> acc_kernel_grad(
      regions[5], task->regions[5], FID_DATA, ctx, runtime,
      true/*readOutput*/);
  TensorAccessorW<float, 1> acc_bias_grad(
      regions[6], task->regions[6], FID_DATA, ctx, runtime,
      true/*readOutput*/);
  // make sure the sizes match
  assert(acc_output.rect.volume() == out_dim * batch_size);
  assert(acc_output_grad.rect.volume() == out_dim * batch_size);
  assert(acc_kernel.rect.volume() == in_dim * out_dim);
  assert(acc_kernel_grad.rect.volume() == in_dim * out_dim);
  assert(acc_bias_grad.rect.volume() == out_dim);
  hipEvent_t t_start, t_end;
  if (linear->profiling) {
    hipEventCreate(&t_start);
    hipEventCreate(&t_end);
    hipEventRecord(t_start);
  }
#ifndef DISABLE_LEGION_CUDA_HIJACK
  //hipStream_t stream;
  //checkCUDA(hipStreamCreate(&stream));
  //checkCUDA(hipblasSetStream(m->handle.blas, stream));
  //checkCUDNN(hipdnnSetStream(m->handle.dnn, stream));
#endif
  if (linear->activation == AC_MODE_RELU) {
    reluBackward<<<GET_BLOCKS(acc_output.rect.volume()), CUDA_NUM_THREADS>>>(
        acc_output_grad.ptr, acc_output.ptr, acc_output.rect.volume());
  } else if (linear->activation == AC_MODE_SIGMOID) {
    sigmoid_backward<<<GET_BLOCKS(acc_output.rect.volume()), CUDA_NUM_THREADS>>>(
        acc_output_grad.ptr, acc_output.ptr, acc_output.rect.volume());
  } else {
    // TODO: only support relu and sigmoid for now
    assert(linear->activation == AC_MODE_NONE);
  }
  // Compute weight gradiant
  // NOTE: we use alpha=1 for kernel_grad to accumulate gradients
  checkCUDA(hipblasSgemm(m->handle.blas, HIPBLAS_OP_N, HIPBLAS_OP_T,
                        in_dim, out_dim, batch_size,
                        &alpha, acc_input.ptr, in_dim,
                        acc_output_grad.ptr, out_dim,
                        &alpha, acc_kernel_grad.ptr, in_dim));
  // Compute bias gradiant
  // NOTE: we use alpha=1 for bias_grad to accumulate gradients
  checkCUDA(hipblasSgemv(m->handle.blas, HIPBLAS_OP_N,
                        out_dim, batch_size,
                        &alpha, acc_output_grad.ptr, out_dim,
                        m->one_ptr, 1,
                        &alpha, acc_bias_grad.ptr, 1));
  // Compute data gradiant
  // NOTE: we use alpha=1 for input_grad to accumulate gradients
  checkCUDA(hipblasSgemm(m->handle.blas, HIPBLAS_OP_N, HIPBLAS_OP_N,
                        in_dim, batch_size, out_dim,
                        &alpha, acc_kernel.ptr, in_dim,
                        acc_output_grad.ptr, out_dim,
                        &alpha, input_grad, in_dim));
  if (linear->profiling) {
    hipEventRecord(t_end);
    checkCUDA(hipEventSynchronize(t_end));
    float elapsed = 0;
    checkCUDA(hipEventElapsedTime(&elapsed, t_start, t_end));
    hipEventDestroy(t_start);
    hipEventDestroy(t_end);
    printf("Linear backward time = %.2lfms\n", elapsed);
    //print_tensor<2, float>(acc_output_grad.ptr, acc_output_grad.rect, "[Linear:backward:output_grad]");
    //print_tensor<2, float>(acc_kernel_grad.ptr, acc_kernel_grad.rect, "[Linear:backward:kernel_grad]");
    //print_tensor<1, float>(acc_bias_grad.ptr, acc_bias_grad.rect, "[Linear:backward:bias_grad]");
    //print_tensor<2, float>(input_grad, acc_input.rect, "[Linear:backward:input_grad]");
    checkCUDA(hipDeviceSynchronize());
  }
}

/*
  regions[0](I/O): input_grad
  regions[1](I): replicas
*/
__host__
void Linear::backward2_task(const Task *task,
                            const std::vector<PhysicalRegion> &regions,
                            Context ctx, Runtime *runtime)
{
  float alpha = 1.0f;
  const LinearMeta* m = *((LinearMeta**) task->local_args);
  TensorAccessorW<float, 2> acc_input(
      regions[0], task->regions[0], FID_DATA, ctx, runtime,
      true/*readOutput*/);
  TensorAccessorR<float, 3> acc_replica(
      regions[1], task->regions[1], FID_DATA, ctx, runtime);
  assert(acc_input.rect.hi[0] == acc_replica.rect.hi[0]);
  assert(acc_input.rect.lo[0] == acc_replica.rect.lo[0]);
  assert(acc_input.rect.hi[1] == acc_replica.rect.hi[1]);
  assert(acc_input.rect.lo[1] == acc_replica.rect.lo[1]);
  hipStream_t stream;
  checkCUDA(hipStreamCreate(&stream));
  checkCUDA(hipblasSetStream(m->handle.blas, stream));
  checkCUDNN(hipdnnSetStream(m->handle.dnn, stream));
  int num_replica = acc_replica.rect.hi[2] - acc_replica.rect.lo[2] + 1;
  const float *replica_ptr = acc_replica.ptr;
  for (int i = 1; i < num_replica; i++) {
    checkCUDA(hipblasSaxpy(m->handle.blas, acc_input.rect.volume(),
                          &alpha, replica_ptr, 1, acc_input.ptr, 1));
    replica_ptr += acc_input.rect.volume();
  }
}

void Linear::backward(const FFModel& ff)
{
  ArgumentMap argmap;
  Context ctx = ff.config.lg_ctx;
  Runtime* runtime = ff.config.lg_hlr;
  Rect<2> rect = runtime->get_index_space_domain(ctx, task_is);
  int idx = 0;
  for (PointInRectIterator<2> it(rect); it(); it++) {
    OpMeta* mp = meta[idx++];
    argmap.set_point(*it, TaskArgument(&mp, sizeof(OpMeta*)));
  }
  {
    IndexLauncher launcher(LINEAR_BWD_TASK_ID, task_is,
                           TaskArgument(this, sizeof(Linear)), argmap,
                           Predicate::TRUE_PRED, false/*must*/, 0/*mapper_id*/,
                           FFConfig::get_hash_id(std::string(name)));
    // regions[0](I): input
    launcher.add_region_requirement(
        RegionRequirement(input_lps[0], 0/*projection id*/,
                          READ_ONLY, EXCLUSIVE, inputs[0].region));
    launcher.add_field(0, FID_DATA);
    // regions[1](I/O): replica_grad 
    if (replica.region_grad != LogicalRegion::NO_REGION) {
      launcher.add_region_requirement(
          RegionRequirement(replica.part_grad, 0/*projection id*/,
                            READ_WRITE, EXCLUSIVE, replica.region_grad));
      launcher.add_field(1, FID_DATA);
    } else {
      launcher.add_region_requirement(
          RegionRequirement(input_grad_lps[0], 0/*projection id*/,
                            READ_WRITE, EXCLUSIVE, inputs[0].region_grad));
      launcher.add_field(1, FID_DATA);
    }
    // regions[2](I): output
    launcher.add_region_requirement(
        RegionRequirement(outputs[0].part, 0/*projection id*/,
                          READ_ONLY, EXCLUSIVE, outputs[0].region));
    launcher.add_field(2, FID_DATA);
    // regions[3](I/O): output_grad
    launcher.add_region_requirement(
        RegionRequirement(outputs[0].part_grad, 0/*projection id*/,
                          READ_WRITE, EXCLUSIVE, outputs[0].region_grad));
    launcher.add_field(3, FID_DATA);
    // regions[4](I): filter
    launcher.add_region_requirement(
        RegionRequirement(weights[0].part, 0/*projection id*/,
                          READ_ONLY, EXCLUSIVE, weights[0].region));
    launcher.add_field(4, FID_DATA);
    // regions[5](I/O): filter_grad
    launcher.add_region_requirement(
        RegionRequirement(weights[0].part_grad, 0/*projection id*/,
                          READ_WRITE, EXCLUSIVE, weights[0].region_grad));
    launcher.add_field(5, FID_DATA);
    // regions[6](I/O): bias_grad
    launcher.add_region_requirement(
        RegionRequirement(weights[1].part_grad, 0/*projection id*/,
                          READ_WRITE, EXCLUSIVE, weights[1].region_grad));
    launcher.add_field(6, FID_DATA);
    runtime->execute_index_space(ctx, launcher);
  }
  if (replica.region_grad != LogicalRegion::NO_REGION) {
    // We aggregate parameters from replica tensor to input tensor
    // Note we use input's task_is to reduce extra data transfers
    Rect<2> input_rect = runtime->get_index_partition_color_space(
      ctx, inputs[0].part_grad.get_index_partition());
    IndexSpaceT<2> input_task_is = IndexSpaceT<2>(ff.get_task_is(input_rect));
    IndexLauncher launcher(LINEAR_BWD2_TASK_ID, task_is,
                           TaskArgument(this, sizeof(Linear)), argmap,
                           Predicate::TRUE_PRED, false/*must*/, 0/*mapper_id*/,
                           FFConfig::get_hash_id(std::string(name)));
    launcher.add_region_requirement(
        RegionRequirement(input_grad_lps[0], 0/*projection id*/,
                          READ_WRITE, EXCLUSIVE, inputs[0].region_grad));
    launcher.add_field(0, FID_DATA);
    // Note that replica.part save's a partition of replica.region_grad
    launcher.add_region_requirement(
        RegionRequirement(replica.part, 0/*partition id*/,
                          READ_ONLY, EXCLUSIVE, replica.region_grad));
    launcher.add_field(1, FID_DATA);
    runtime->execute_index_space(ctx, launcher);
  }
}

/*
__host__
Parameter* Linear::get_parameter(int index)
{
  if (index == 0) {
    return &weights[0];
  } else if (index == 1){
    return &weights[1];
  } else {
    assert(0);
    return NULL;
  }
}
*/

__host__
void Linear::print_layer(const FFModel& ff)
{
  printf("linear layer\n");  
  Context ctx = ff.config.lg_ctx;
  Runtime* runtime = ff.config.lg_hlr;

  RegionRequirement kernel_req(weights[0].region, READ_WRITE, EXCLUSIVE, weights[0].region);
  kernel_req.add_field(FID_DATA);
  InlineLauncher kernel_launcher(kernel_req);
  PhysicalRegion kernel_region = runtime->map_region(ctx, kernel_launcher);
  kernel_region.wait_until_valid();
  
  RegionRequirement bias_req(weights[1].region, READ_WRITE, EXCLUSIVE, weights[1].region);
  bias_req.add_field(FID_DATA);
  InlineLauncher bias_launcher(bias_req);
  PhysicalRegion bias_region = runtime->map_region(ctx, bias_launcher);
  bias_region.wait_until_valid();
  
  TensorAccessorW<float, 2> acc_kernel(kernel_region, kernel_req, FID_DATA, ctx, runtime, true);
  TensorAccessorW<float, 1> acc_bias(bias_region, bias_req, FID_DATA, ctx, runtime, true);
  
  const float *kernel_ptr = acc_kernel.ptr;
  const float *bias_ptr = acc_bias.ptr;
  
  size_t kernel_size = acc_kernel.rect.volume();
  int kernel_dim1 = acc_kernel.rect.hi[0] - acc_kernel.rect.lo[0] + 1;
  int kernel_dim2 = acc_kernel.rect.hi[1] - acc_kernel.rect.lo[1] + 1;
  size_t bias_size = acc_bias.rect.volume();
  printf("kernel, %p, %d, [%d, %d]\n", kernel_ptr, kernel_size, kernel_dim1, kernel_dim2);
  printf("bias, %p, %d\n", bias_ptr, bias_size);

  
  for (int i = 0; i < bias_size; i++) {
    printf("%f ", bias_ptr[i]);
  }
  printf("\n");
  
  for (int i = 0; i < kernel_size; i++) {
    printf("%f ", kernel_ptr[i]);
  }
  printf("\n");
  
  runtime->unmap_region(ctx, kernel_region);
  runtime->unmap_region(ctx, bias_region);

}

